#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include "parameter.cuh"


typedef hiprandStatePhilox4_32_10_t myCurandState_t; 


// #define DEBUG


#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

#define TOTAL (SIZE * SIZE)

#define SRAND_VALUE 200

#define CONFLICT_LIST_LENGTH 10

const int agentTypeOneNumber = agentNumber / 2;
const int agentTypeTwoNumber = agentNumber - agentTypeOneNumber;
const int happinessThreshold = 5;
const int numThreadsPerBlock = 256;
const int limitedNeighbourhood = 3;

void printOutput(int [SIZE+2][SIZE+2]);
void initPos(int grid [SIZE+2][SIZE+2]);
int random_location();


__device__ unsigned int numberConflict = 0;
__device__ unsigned int numberMoveable = 0;


__device__ int getnextrand(myCurandState_t *state){

	int number = (1 + (int)(hiprand_uniform(state)*(SIZE)));
	return number;
}

__global__ void initCurand(myCurandState_t state[][SIZE+2], unsigned long seed){
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	int idy=blockIdx.y*blockDim.y+threadIdx.y;
	hiprand_init( 0 ,idx*(SIZE+2)+idy+10, 0, &state[idx][idy]);

}



__global__ void compute(int grid[][SIZE+2], int new_grid[][SIZE+2], int temp_grid[][SIZE+2]){
	
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	int idy=blockIdx.y*blockDim.y+threadIdx.y;
	int sameTypeCount=0;
	int current_priority = idx*(SIZE+2)+idy;

	if(grid[idx][idy] != 0){
		int currentType = grid[idx][idy];

		if(grid[idx-1][idy-1] == currentType){
			sameTypeCount += 1;
		}

		if(grid[idx-1][idy] == currentType){
			sameTypeCount += 1;
		}

		if(grid[idx-1][idy+1] == currentType){
			sameTypeCount += 1;
		}

		if(grid[idx][idy-1] == currentType){
			sameTypeCount += 1;
		}

		if(grid[idx][idy+1] == currentType){
			sameTypeCount += 1;
		}

		if(grid[idx+1][idy-1] == currentType){
			sameTypeCount += 1;
		}

		if(grid[idx+1][idy] == currentType){
			sameTypeCount += 1;
		}

		if(grid[idx+1][idy+1] == currentType){
			sameTypeCount += 1;
		}

		if(sameTypeCount < happinessThreshold){

			
			temp_grid[idx][idy] =  current_priority;
		}
	}
	

}



__global__ void prepareNewGrid (int temp_grid[][SIZE+2], int new_grid[][SIZE+2]){
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	int idy=blockIdx.y*blockDim.y+threadIdx.y;

	if(temp_grid[idx][idy] != 0){
		new_grid[idx][idy] = 0;

	}
}




__device__ bool agentsLeft;

__global__ void assign_ (myCurandState_t state[][SIZE+2],int grid[][SIZE+2], int new_grid[][SIZE+2], int temp_grid[][SIZE+2],int move_grid[][SIZE+2][CONFLICT_LIST_LENGTH], int move_grid_counters[][SIZE+2], int rowAndColumn[][SIZE+2]){

	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	int idy=blockIdx.y*blockDim.y+threadIdx.y;
	int current_priority = idx*(SIZE+2)+idy;	
	int row = 0;
	int column = 0;
	int old_value;

    bool locallyfailed = false;

	if(temp_grid[idx][idy] != 0 ){

        int loopCounter = 0;
        do {
            if(!locallyfailed) {
                int randomRow = (getnextrand(&state[idx][idy]) % limitedNeighbourhood) - (limitedNeighbourhood / 2);
                row = idx + randomRow;

                int randomColumn = (getnextrand(&state[idx][idy]) % limitedNeighbourhood) - (limitedNeighbourhood / 2);

                column = idy + randomColumn;

            } else {
             
    			row = getnextrand(&state[idx][idy]);
    			column = getnextrand(&state[idx][idy]);
            }

			if(row>=1 && row <=SIZE && column>=1 && column<=SIZE && new_grid[row][column] == 0 ){
                
				old_value = atomicAdd(&move_grid_counters[row][column], 1);
                move_grid[row][column][old_value] = current_priority;

                return;

			}
            loopCounter++;
            if(loopCounter > 10){
                locallyfailed = true;
            }

		} while(true);
	}
}


__global__ void updateTonew (int grid[][SIZE+2], int new_grid[][SIZE+2],int temp_grid[][SIZE+2],int move_grid[][SIZE+2][CONFLICT_LIST_LENGTH], int move_grid_counters[][SIZE+2], int rowAndColumn[][SIZE+2], myCurandState_t state[][SIZE + 2]){
	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	int idy=blockIdx.y*blockDim.y+threadIdx.y;
	
    int num_agents = move_grid_counters[idx][idy];

    if(!num_agents)
        return;

    int priority = 0;
    if(num_agents == 1)
        priority = move_grid[idx][idy][0];

    if(num_agents > 1){
        int r = getnextrand(&state[idx][idy]) % num_agents;
        priority = move_grid[idx][idy][r];
        agentsLeft = true;
    }

   	int source_row = priority / (SIZE + 2);
   	int source_col = priority % (SIZE + 2);
   	new_grid[idx][idy] = grid[source_row][source_col];
   	temp_grid[source_row][source_col] = 0;
}

__global__ void newTogrid (int grid[][SIZE+2], int new_grid[][SIZE+2]){

	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	int idy=blockIdx.y*blockDim.y+threadIdx.y;
	grid[idx][idy] = new_grid[idx][idy];
	

}

__global__ void clearMoveGrid (int move_grid_counters[][SIZE+2]){

	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	int idy=blockIdx.y*blockDim.y+threadIdx.y;
	move_grid_counters[idx][idy] = 0;


}

__global__ void update ( int temp_grid[][SIZE+2],int move_grid_counters[][SIZE+2]){

	int idx=blockIdx.x*blockDim.x+threadIdx.x;
	int idy=blockIdx.y*blockDim.y+threadIdx.y;
	temp_grid[idx][idy] = 0;
	move_grid_counters[idx][idy] = 0;
	

}


void checkNumber(int grid [SIZE+2][SIZE+2]){

	int agentTypeOne = 0;
	int agentTypeTwo = 0;


	for(int i=0; i<SIZE+2; i++){
		for(int j=0; j<SIZE+2; j++){
			if(grid[i][j] == 1){
				agentTypeOne +=1;	

			}
			else if(grid[i][j] == 2){
				agentTypeTwo += 1;

			}
		}

	}

	printf("Type One %d, Type Two %d\n",agentTypeOne, agentTypeTwo);




}


int host_grid[SIZE+2][SIZE+2]; 

int main(int argc, char* argv[])
{

	hipDeviceSetLimit(hipLimitPrintfFifoSize,  10*1024*1024);


 	struct timespec start, stop;
    	double accum;
	int (*device_grid)[SIZE + 2];
	int (*device_newGrid)[SIZE + 2];
	int (*device_moveGrid)[SIZE + 2][CONFLICT_LIST_LENGTH];
	int (*device_moveGridCounters)[SIZE + 2];

	int (*device_tempGrid)[SIZE + 2];

 	int (*device_rowAndColumn)[SIZE + 2];

	srand(SRAND_VALUE);

	size_t bytes = sizeof(int)*(SIZE + 2)*(SIZE + 2);
	myCurandState_t (*devState)[SIZE + 2];
	bool agentsRemain = false;
	
	hipMalloc((void**)&devState, (SIZE+2)*(SIZE+2) * sizeof(myCurandState_t));

	hipMalloc((void**)&device_grid, bytes);
	hipMalloc((void**)&device_newGrid, bytes);
	hipMalloc((void**)&device_tempGrid, bytes);
	hipMalloc((void**)&device_moveGrid, bytes * CONFLICT_LIST_LENGTH);
	hipMalloc((void**)&device_moveGridCounters, bytes);

	hipMalloc((void**)&device_rowAndColumn, bytes);


	int blockSizePerDim = sqrt(numThreadsPerBlock);
	int gridSizePerDim = (SIZE + 2) / blockSizePerDim;

	dim3 blockSize(blockSizePerDim, blockSizePerDim, 1);
	dim3 gridSize(gridSizePerDim, gridSizePerDim, 1);

	initCurand<<<gridSize , blockSize>>>(devState, 1);
	for (int i=0; i<(SIZE+2); i++){
		for (int j=0; j<SIZE+2; j++){
			host_grid[i][j] = 0;
		}
	}


	
	initPos(host_grid);
	// printOutput(host_grid);

	hipMemcpy(device_grid,host_grid,bytes,hipMemcpyHostToDevice);
	hipMemcpy(device_newGrid,host_grid,bytes,hipMemcpyHostToDevice);
	

	newTogrid << <gridSize, blockSize >> >(device_grid, device_newGrid);

	update << <gridSize, blockSize >> >(device_tempGrid,device_moveGridCounters);
	if( clock_gettime( CLOCK_REALTIME, &start) == -1 ) {
    	   perror( "clock gettime" );
   	   exit( EXIT_FAILURE );
   	 }
	
	int numRoundsTotal = atoi(argv[1]);
	for(int i=0; i<numRoundsTotal; i++){


		compute << <gridSize, blockSize >> >(device_grid, device_newGrid,device_tempGrid);

		#ifdef DEBUG
			hipDeviceSynchronize();
			cudaCheckError();
		#endif
		

		
		prepareNewGrid<<<gridSize, blockSize>>>(device_tempGrid,device_newGrid);


		#ifdef DEBUG
			hipDeviceSynchronize();
			cudaCheckError();
		#endif


		do{

			agentsRemain = false;
			hipMemcpyToSymbol(HIP_SYMBOL(agentsLeft),&agentsRemain,sizeof(bool),0,hipMemcpyHostToDevice);

			assign_ << <gridSize, blockSize >> >(devState,device_grid, device_newGrid,device_tempGrid,device_moveGrid, device_moveGridCounters, device_rowAndColumn);
			
			updateTonew << <gridSize, blockSize >> >(device_grid, device_newGrid,device_tempGrid,device_moveGrid,device_moveGridCounters,device_rowAndColumn, devState);

			clearMoveGrid<<<gridSize, blockSize >>>(device_moveGridCounters);
			hipMemcpyFromSymbol(&agentsRemain,HIP_SYMBOL(agentsLeft),sizeof(bool),0, hipMemcpyDeviceToHost);

		}while(agentsRemain == true);


		newTogrid << <gridSize, blockSize >> >(device_grid, device_newGrid);

	
		update << <gridSize, blockSize >> >(device_tempGrid,device_moveGridCounters);
		


	}
	hipDeviceSynchronize();



	
	if( clock_gettime( CLOCK_REALTIME, &stop) == -1 ) {
    	   perror( "clock gettime" );
   	   exit( EXIT_FAILURE );
   	 }

	accum = ( stop.tv_sec - start.tv_sec ) * 1e6
          + ( stop.tv_nsec - start.tv_nsec ) / 1e3;
	
    	printf( "%.1f Time is %.5f s \n", float(OCCUPANCY), accum / 1e6);
	hipMemcpy(host_grid, device_grid, bytes, hipMemcpyDeviceToHost);
	//printOutput(host_grid);
	//checkNumber(host_grid);
	hipFree(device_grid);
	hipFree(device_newGrid);
	hipFree(device_tempGrid);
	hipFree(devState);
	hipFree(device_rowAndColumn);

	return 0;


}



void printOutput(int grid [SIZE+2][SIZE+2]  ){ //output grid from 1 t o SIZE+1
 	
	for (int i=0; i<SIZE+2; i++){
		for (int j=0; j<SIZE+2; j++){
			printf("%d ",grid[i][j]);
		//if(i%SIZE)
		}		
		printf("\n");
	}
	printf("\n");
}



void initPos(int grid [SIZE+2][SIZE+2]){  //assign type 1 and 2 to grid randomly
	int row;
	int column;
	for(int i=0; i<agentTypeOneNumber; i++){
		do{
			row = random_location();
			column = random_location();
		}while(grid[row][column] != 0);
		
		grid[row][column] = 1;	
	}

	for(int i=0; i<agentTypeTwoNumber; i++){
		do{
			row = random_location();
			column = random_location();
		}while(grid[row][column] != 0);
		
		grid[row][column] = 2;	
	}




}


int random_location() { //generate a random number from 1 to SIZE

	int r;

	r = rand();

	return (r % (SIZE) +1 );


}










